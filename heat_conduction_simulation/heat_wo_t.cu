#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../common/cpu_anim.h"
#include "../common/book.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

// these exist on the GPU side
// 这些变量将位与GPU上
/*
texture<float>  texConstSrc;
texture<float>  texIn;
texture<float>  texOut;
*/


// this kernel takes in a 2-d array of floats
// it updates the value-of-interest by a scaled value based
// on itself and its nearest neighbors
//给定一个输入格网，据等式7.2中的更新公式计算输出温度格网。
// this kernel takes in a 2-d array of floats
// it updates the value-of-interest by a scaled value based
// on itself and its nearest neighbors
//给定一个输入格网，据等式7.2中的更新公式计算输出温度格网。
__global__ void blend_kernel( float *in,
                              float *dst ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset - 1;
    int right = offset + 1;
    if (x == 0)   left++;
    if (x == DIM-1) right--; 

    int top = offset - DIM;
    int bottom = offset + DIM;
    if (y == 0)   top += DIM;
    if (y == DIM-1) bottom -= DIM;

    float   t, l, c, r, b;

    t = in[top];
    l = in[left];
    c = in[offset];
    r = in[right];
    b = in[bottom];

    
    dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
    __syncthreads();
}

// NOTE - texOffsetConstSrc could either be passed as a
// parameter to this function, or passed in __constant__ memory
// if we declared it as a global above, it would be
// a parameter here: 
//给定一个包含初始输入温度的格网，将其中作为热源的单元温度值复制到格网相应单元中。这将覆盖这些大院之前计算出的温度。
// __global__ void copy_const_kernel( float *iptr,
//                                    size_t texOffset )
__global__ void copy_const_kernel( float *iptr, float *constSrc) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (constSrc[offset] != 0)
        iptr[offset] = constSrc[offset];
    __syncthreads();
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *output_bitmap;
    float           *dev_inSrc;
    float           *dev_outSrc;
    float           *dev_constSrc;
    CPUAnimBitmap  *bitmap;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
    FILE           *fp;
    int             imageSize;
};

void anim_gpu( DataBlock *d, int times ) {
    HANDLE_ERROR( hipEventRecord( d->start, 0 ) );
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
    CPUAnimBitmap  *bitmap = d->bitmap;

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    volatile bool dstOut = true;
    float *constSrc = d->dev_constSrc;
    float   *in, *out, *tmp;
    in  = d->dev_inSrc;
    out = d->dev_outSrc;
    for (int i=0; i<90 * times; i++) {
        copy_const_kernel<<<blocks,threads>>>( in, constSrc );
        blend_kernel<<<blocks,threads>>>( in, out );
        tmp = out;
        out = in;
        in = tmp;
    }
    float_to_color<<<blocks,threads>>>( d->output_bitmap,
                                        out );
    HANDLE_ERROR( hipEventRecord( d->stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( d->stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        d->start, d->stop ) );
    HANDLE_ERROR( hipMemcpy( bitmap->get_ptr(),
                              d->output_bitmap,
                              bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );    
    d->totalTime += elapsedTime;
    ++d->frames;
    printf( "Average Time per frame:  %3.1f ms\n",
            d->totalTime/times);
    /*
    if (ticks == 100){
        fwrite(bitmap->pixels, d->imageSize , 1, d->fp );
    }
    */
}

// clean up memory allocated on the GPU
void anim_exit( DataBlock *d ) {
    /*
    hipUnbindTexture( texIn );
    hipUnbindTexture( texOut );
    hipUnbindTexture( texConstSrc );
    */
    HANDLE_ERROR( hipFree( d->dev_inSrc ) );
    HANDLE_ERROR( hipFree( d->dev_outSrc ) );
    HANDLE_ERROR( hipFree( d->dev_constSrc ) );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
    fclose(d->fp);
}


int main( void ) {
    DataBlock   data;
    data.fp = fopen( "file.txt" , "w" );
    CPUAnimBitmap bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    HANDLE_ERROR( hipEventCreate( &data.start ) );
    HANDLE_ERROR( hipEventCreate( &data.stop ) );

    data.imageSize = bitmap.image_size();

    HANDLE_ERROR( hipMalloc( (void**)&data.output_bitmap,
                               data.imageSize ) );

    // assume float == 4 chars in size (ie rgba)
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_inSrc,
                              data.imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_outSrc,
                              data.imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_constSrc,
                              data.imageSize ) );
    /*
    HANDLE_ERROR( hipBindTexture( NULL, texConstSrc,
                                   data.dev_constSrc,
                                   data.imageSize ) );

    HANDLE_ERROR( hipBindTexture( NULL, texIn,
                                   data.dev_inSrc,
                                   data.imageSize ) );

    HANDLE_ERROR( hipBindTexture( NULL, texOut,
                                   data.dev_outSrc,
                                   data.imageSize ) );
    */

    // intialize the constant data
    float *temp = (float*)malloc( data.imageSize );
    for (int i=0; i<DIM*DIM; i++) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x>300) && (x<600) && (y>310) && (y<601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM*100+100] = (MAX_TEMP + MIN_TEMP)/2;
    temp[DIM*700+100] = MIN_TEMP;
    temp[DIM*300+300] = MIN_TEMP;
    temp[DIM*200+700] = MIN_TEMP;
    for (int y=800; y<900; y++) {
        for (int x=400; x<500; x++) {
            temp[x+y*DIM] = MIN_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy( data.dev_constSrc, temp,
                              data.imageSize,
                              hipMemcpyHostToDevice ) );    
    // initialize the input data
    for (int y=800; y<DIM; y++) {
        for (int x=0; x<200; x++) {
            temp[x+y*DIM] = MAX_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy( data.dev_inSrc, temp,
                              data.imageSize,
                              hipMemcpyHostToDevice ) );
    free( temp );
    anim_gpu(&data, 500);
    fwrite(data.bitmap->pixels, data.imageSize , 1, data.fp); //write calculate result
    anim_exit(&data);
}
